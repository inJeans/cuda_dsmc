#include "hip/hip_runtime.h"
/** \file
 *  \brief Functions necessary for generating a thermal distribution
 *
 *  More detailed description
 *  Copyright 2015 Christopher Watkins
 */

#include <assert.h>
#include <math.h>
#include <stdint.h>

#include "random_numbers.cuh"

/** \fn __host__ void cu_initialise_rng_states(int n_states,
                                               hiprandState *state) 
 *  \brief Fills the array states with n_state seeds for the rng
 *  \param n_seeds Number of rng seeds required.
 *  \param *state Pointer to the an array of length n_seeds.
 *  \exception not yet.
 *  \return void
*/

__host__ void cu_initialise_rng_states(int n_states,
                                       hiprandState *state) {
    LOGF(INFO, "\nCalculating optimal launch configuration for the state intialisation kernel.\n");
    int block_size = 0;
    int min_grid_size = 0;
    int grid_size = 0;
    hipOccupancyMaxPotentialBlockSize(&min_grid_size,
                                       &block_size,
                                       (const void *) g_initialise_rng_states,
                                       0,
                                       n_states);
    grid_size = (n_states + block_size - 1) / block_size;

    LOGF(INFO, "\nLaunch config set as <<<%i,%i>>>\n", grid_size, block_size);
    g_initialise_rng_states<<<grid_size,
                              block_size>>>
                           (n_states,
                            state);
    return;
}

/** \fn __global__ void setup_kernel(int n_states,
 *                                   hiprandState *state) 
 *  \brief Fills the array states with n_state seeds for the rng
 *  \param n_seeds Number of rng seeds required.
 *  \param *state Pointer to the an array of length n_seeds.
 *  \exception not yet.
 *  \return void
*/

__global__ void g_initialise_rng_states(int n_states,
                                        hiprandState *state) {
    for (int id = blockIdx.x * blockDim.x + threadIdx.x;
         id < n_states;
         id += blockDim.x * gridDim.x) {
        /* Each thread gets same seed, a different sequence number, 
           no offset */
        hiprand_init(1234, id, 0, &state[id]);
    }
    return;
}

/** \fn __device__ double3 gaussian_point(double mean,
 *                                        double std,
 *                                        hiprandState *seed) 
 *  \brief Generates a double3 where each element is normally distributed
 *  with mean and std as the mean and standard deviation respectively
 *  \param mean Gaussian mean
 *  \param std standard deviation
 *  \param *seed seed for the rng
 *  \exception not yet.
 *  \return a gaussian distributed point in cartesian space
*/

__device__ double3 gaussian_point(double mean,
                                  double std,
                                  hiprandState *state) {
    double3 p = make_double3(0., 0., 0.);
    p.x = hiprand_normal(state);
    p.y = hiprand_normal(state);
    p.z = hiprand_normal(state);

    return p;
}
