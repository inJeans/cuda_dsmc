#include "hip/hip_runtime.h"
/** \file
 *  \brief Functions necessary for generating a thermal distribution
 *
 *  More detailed description
 *  Copyright 2015 Christopher Watkins
 */

#include "distribution_generation.cuh"

__constant__ double d_gs   =  0.5;      // Gyromagnetic ratio
__constant__ double d_MF   = -1.0;      // Magnetic quantum number
__constant__ double d_muB  = 9.27400915e-24;  // Bohr magneton
__constant__ double d_mass = 1.443160648e-25;// 87Rb mass
__constant__ double d_pi   = 3.14159265;    // Pi
__constant__ double d_a    = 5.3e-9;      // Constant cross-section formula
__constant__ double d_kB   = 1.3806503e-23; // Boltzmann's Constant
__constant__ double d_hbar = 1.05457148e-34;  // hbar

/** \fn __host__ void cu_generate_thermal_velocities(int num_atoms,
 *                                                   double temp,
 *                                                   hiprandState *state,
                                                     double3 *vel) 
 *  \brief Calls the function to fill an array of thermal velocties at
 *  temperature temp.
 *  \param mean Gaussian mean
 *  \exception not yet.
 *  \return a gaussian distributed point in cartesian space
*/

__host__ void cu_generate_thermal_velocities(int num_atoms,
                                             double temp,
                                             hiprandState *state,
                                             double3 *vel) {
    int block_size = 0;
    int min_grid_size = 0;
    int grid_size = 0;
    hipOccupancyMaxPotentialBlockSize(&min_grid_size,
                                       &block_size,
                                       (const void *) g_generate_thermal_velocities,
                                       0,
                                       num_atoms);
    grid_size = (num_atoms + block_size - 1) / block_size;

    g_generate_thermal_velocities<<<grid_size,
                                    block_size>>>
                                 (num_atoms,
                                  temp,
                                  state,
                                  vel);  

    return;
}

/** \fn __global__ void g_generate_thermal_velocities(int num_atoms,
 *                                                    double temp,
 *                                                    hiprandState *state,
 *                                                    double3 *vel) 
 *  \brief description
 *  \param num_atoms Total number of atoms in the gas.
 *  \param temp Temperature of the gas (in Kelvin).
 *  \param *seed Pointer to an array of seeds for the random number generator
 *  \param *vel Pointer to an output array of length num_atoms for storing
    the gas velocities.
 *  \exception not yet.
 *  \return void
*/

__global__ void g_generate_thermal_velocities(int num_atoms,
                                              double temp,
                                              hiprandState *state,
                                              double3 *vel) {
    for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
         atom < num_atoms;
         atom += blockDim.x * gridDim.x) {
        vel[atom] = d_thermal_vel(temp,
                                  &state[atom]);
    }

    return;
}

/** \fn __host__ __device__ double3 d_thermal_vel(double temp,
 *                                                hiprandState *state) 
 *  \brief description
 *  \param temp Temperature of the gas (in Kelvin).
 *  \param *seed Pointer to a seed for the random number generator.
 *  \exception not yet.
 *  \return a gaussian distributed point in cartesian space with the standard
    deviation expected for a thermal gas
*/

__device__ double3 d_thermal_vel(double temp,
                                 hiprandState *state) {
    double V = sqrt(d_kB * temp / d_mass);
    double3 vel = gaussian_point(0,
                                 V,
                                 state);
    return vel;
}
