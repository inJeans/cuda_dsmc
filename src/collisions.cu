#include "hip/hip_runtime.h"
/** \file
 *  \brief Functions necessary for colliding a distribution of atoms on device
 *
 *  More detailed description
 *  Copyright 2015 Christopher Watkins
 */

#include "collisions.cuh"

#include "declare_device_constants.cuh"

/** \fn __host__ void cu_index_atoms(int num_atoms,
 *                                   double3 *pos,
 *                                   int *cell_id) 
 *  \brief Calls the `__global__` function to update an `int` device array with
 *  cell_ids based on the atoms position and the maximum cell width.
 *  \param num_atoms Number of atoms in the thermal gas.
 *  \param *pos Pointer to a `double3` device array of length
 *  `num_atoms` containing the positions.
 *  \param *cell_id Pointer to an output `int` device array of length
 *  `num_atoms` containing the cell_ids.
 *  \exception not yet.
 *  \return void
*/

__host__ void cu_index_atoms(int num_atoms,
                             double3 *pos,
                             int *cell_id) {
LOGF(DEBUG, "\nCalculating optimal launch configuration for the atom "
                "indexing kernel.\n");
    int block_size = 0;
    int min_grid_size = 0;
    int grid_size = 0;
    hipOccupancyMaxPotentialBlockSize(&min_grid_size,
                                       &block_size,
                                       (const void *) g_index_atoms,
                                       0,
                                       num_atoms);
    grid_size = (num_atoms + block_size - 1) / block_size;
    LOGF(DEBUG, "\nLaunch config set as <<<%i,%i>>>\n",
                grid_size, block_size);
    g_index_atoms<<<grid_size,
                    block_size>>>
                   (num_atoms,
                    pos,
                    cell_id);

    return;
}

/** \fn __global__ void g_index_atoms(int num_atoms,
 *                                    double3 *pos,
 *                                    int *cell_id) 
 *  \brief `__global__` function to update an `int` device array with
 *  cell_ids based on the atoms position and the maximum cell width.
 *  \param num_atoms Number of atoms in the thermal gas.
 *  \param *pos Pointer to a `double3` device array of length
 *  `num_atoms` containing the positions.
 *  \param *cell_id Pointer to an output `int` device array of length
 *  `num_atoms` containing the cell_ids.
 *  \exception not yet.
 *  \return void
*/

__global__ void g_index_atoms(int num_atoms,
                              double3 *pos,
                              int *cell_id) {
    for (int atom = 0; atom < num_atoms; ++atom) {
        cell_id[atom] = d_update_atom_cell_id(pos[atom]);
    }

    return;
}

/** \fn __device__ int d_update_atom_cell_id(double3 pos) 
 *  \brief Calls the function to calculate the cell ID of an atom based on its
 *  current position. Cell IDs are counted from the negative end of each
 *  cartesian direction, first along `x`, then along `y` and finally along `z`.
 *  \param pos The position of the atom.
 *  \exception not yet.
 *  \return cell_id An integer containing the cell ID of the atom.
*/

__device__ int d_update_atom_cell_id(double3 pos) {
    int cell_id = 0;

    int3 cell_index = d_atom_cell_index(pos);
    cell_id = d_atom_cell_id(cell_index);

    return cell_id;
}

/** \fn __device__ int3 d_atom_cell_index(double3 pos) 
 *  \brief Calls the function to calculate the individual cell indicies for each
 *  cartesian direction based on an atoms current position.
 *  \param pos The position of the atom.
 *  \exception not yet.
 *  \return cell_index An `int3` containing the individual cell indicies for
 *  each cartesian direction.
*/

__device__ int3 d_atom_cell_index(double3 pos) {
    int3 cell_index = make_int3(0, 0, 0);

    // NOTE: Computer scientists may have a problem with this typecast since,
    //       integers cannot store the same maximum number as a float can.
    //       So if we anticipate having more than 2^31 cells, then we need
    //       to do something smarter here.
    cell_index = type_cast_int3(floor((pos - d_grid_min) / d_cell_length));

    return cell_index;
}

/** \fn __device__ int d_atom_cell_id(int3 cell_index) 
 *  \brief Calls the function to combine the individual cell indicies for each
 *  cartesian direction into the singal global `cell_id`.
 *  \param cell_index The cartesian cell indices of the atom.
 *  \exception not yet.
 *  \return cell_index An `int` containing the global `cell_id`.
*/

__device__ int d_atom_cell_id(int3 cell_index) {
    int cell_id = 0;

    if (cell_index.x > -1 && cell_index.x < d_num_cells.x &&
        cell_index.y > -1 && cell_index.y < d_num_cells.y &&
        cell_index.z > -1 && cell_index.z < d_num_cells.z) {
        cell_id = cell_index.z*d_num_cells.x*d_num_cells.y +
                  cell_index.y*d_num_cells.x +
                  cell_index.x;
    } else {
        cell_id = d_num_cells.x*d_num_cells.y*d_num_cells.z;
    }

    return cell_id;
}
