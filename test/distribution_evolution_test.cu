#include "hip/hip_runtime.h"
/** \file
 *  \brief Unit tests for the distribution_generation file
 *
 *  More detailed description
 *  Copyright 2015 Christopher Watkins
 */

#include <hip/hip_runtime.h>

#include <float.h>
#include <algorithm>

#define CATCH_CONFIG_MAIN
#include "catch.hpp"

#include "distribution_evolution.hpp"
#include "distribution_generation.hpp"
#include "random_numbers.hpp"
#include "trapping_potential.hpp"
#include "hip/hip_runtime_api.h"
#include "distribution_evolution_tests.hpp"

#include "define_host_constants.hpp"
#include "declare_device_constants.cuh"

SCENARIO("[DEVICE] Acceleration Update", "[d-acc]") {
    GIVEN("A thermal distribution of 5000 positions, help in a quadrupole trap with a Bz = 2.0") {
        int num_test = 5000;

        // Initialise trapping parameters
        trap_geo trap_parameters;
        trap_parameters.Bz = 2.0;
        trap_parameters.B0 = 0.;

        // Initialise rng
        hiprandState *state;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&state),
                                   num_test*sizeof(hiprandState)));
        initialise_rng_states(num_test,
                              state,
                              false);

        // Initialise positions
        double3 *d_test_pos;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_test_pos),
                                   num_test*sizeof(double3)));

        // Generate velocity distribution
        generate_thermal_positions(num_test,
                                   20.e-6,
                                   trap_parameters,
                                   state,
                                   d_test_pos);

        WHEN("The update_atom_accelerations function is called") {
            // Initialise accelerations
            double3 *d_test_acc;
            checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_test_acc),
                                       num_test*sizeof(double3)));

            // Generate accelerations
            update_atom_accelerations(num_test,
                                      trap_parameters,
                                      d_test_pos,
                                      d_test_acc);

            THEN("The result should agree with the kinetic gas theory") {
                // REQUIRE(r >= 0.);
                // REQUIRE(r <= 1.);
            }

            hipFree(d_test_acc);
        }

        hipFree(d_test_pos);
    }
}
