#include "hip/hip_runtime.h"
/** \file
 *  \brief Unit tests for the collisions file
 *
 *  More detailed description
 *  Copyright 2015 Christopher Watkins
 */

#include "collisions_test.cuh"

double fractional_tol = 0.05; 

SCENARIO("[DEVICE] Initialise grid parameters", "[d-initgrid]") {
    GIVEN("A device array of 10 known positions, in a grid with num_cells = {2,3,4}.") {
        double3 pos[10];
        pos[0] = make_double3(0., 0., 0.);
        pos[1] = make_double3(-1., -1., -1.);
        pos[2] = make_double3(1., 1., 1.);
        pos[3] = make_double3(-3., 0., 1.);
        pos[4] = make_double3(10., -3., 4.);
        pos[5] = make_double3(2., 9., -6.);
        pos[6] = make_double3(-8., 15., 7.);
        pos[7] = make_double3(-2., -8., 10.);
        pos[8] = make_double3(1., -2., -10.);
        pos[9] = make_double3(0., 2., 0.);

        double3 *d_pos;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_pos),
                                   10*sizeof(double3)));
        checkCudaErrors(hipMemcpy(d_pos,
                                   pos,
                                   10*sizeof(double3),
                                   hipMemcpyHostToDevice));

        k_num_cells = make_int3(2, 3, 4);

        WHEN("The initialise_grid_params function is called") {
            hipblasHandle_t cublas_handle;
            checkCudaErrors(hipblasCreate(&cublas_handle));
            initialise_grid_params(10,
                                   cublas_handle,
                                   d_pos);

            THEN("Then the device global grid_min = {-10., -15., -10.} ") {
                // Cannot memCpy from constant memory. Need to use a kernel to
                // copy into global memory first.
                double3 *grid_min;
                checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&grid_min),
                                           sizeof(double3)));
                copy_d_grid_min<<<1, 1>>>(grid_min);
                double3 t_grid_min = make_double3(0., 0., 0.);
                checkCudaErrors(hipMemcpy(&t_grid_min,
                                           grid_min,
                                           sizeof(double3),
                                           hipMemcpyDeviceToHost));
                REQUIRE(t_grid_min.x == -10.);
                REQUIRE(t_grid_min.y == -15.);
                REQUIRE(t_grid_min.z == -10.);

                hipFree(grid_min);
            }
            THEN("Then the device global cell_length = {10., 10., 5.} ") {
                // Cannot memCpy from constant memory. Need to use a kernel to
                // copy into global memory first.
                double3 *cell_length;
                checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&cell_length),
                                           sizeof(double3)));
                copy_d_cell_length<<<1, 1>>>(cell_length);
                double3 t_cell_length = make_double3(0., 0., 0.);
                checkCudaErrors(hipMemcpy(&t_cell_length,
                                           cell_length,
                                           sizeof(double3),
                                           hipMemcpyDeviceToHost));
                REQUIRE(t_cell_length.x == 10.);
                REQUIRE(t_cell_length.y == 10.);
                REQUIRE(t_cell_length.z == 5.);

                hipFree(cell_length);
            }

            checkCudaErrors(hipblasDestroy(cublas_handle));
        }

        hipFree(d_pos);
    }
}

SCENARIO("[DEVICE] Index atoms", "[d-index]") {
    GIVEN("An array of 10 known positions, in a grid with num_cells = {2,3,4}.") {
        double3 pos[10];
        pos[0] = make_double3(0., 0., 0.);
        pos[1] = make_double3(-1., -1., -1.);
        pos[2] = make_double3(1., 1., 1.);
        pos[3] = make_double3(-3., 0., 1.);
        pos[4] = make_double3(10., -3., 4.);
        pos[5] = make_double3(2., 9., -6.);
        pos[6] = make_double3(-8., 15., 7.);
        pos[7] = make_double3(-2., -8., 10.);
        pos[8] = make_double3(1., -2., -10.);
        pos[9] = make_double3(0., 2., 0.);
        double3 *d_pos;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_pos),
                                   10*sizeof(double3)));
        checkCudaErrors(hipMemcpy(d_pos,
                                   pos,
                                   10*sizeof(double3),
                                   hipMemcpyHostToDevice));

        k_num_cells = make_int3(2, 3, 4);

        hipblasHandle_t cublas_handle;
        checkCudaErrors(hipblasCreate(&cublas_handle));
        initialise_grid_params(10,
                               cublas_handle,
                               d_pos);

        WHEN("The index_atoms function is called") {
            int *d_cell_id;
            checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_cell_id),
                                       10*sizeof(int)));
            checkCudaErrors(hipMemset(d_cell_id,
                                       0,
                                       10));
            index_atoms(10,
                        d_pos,
                        d_cell_id);

            int test_cell_id[10] = {0};
            checkCudaErrors(hipMemcpy(test_cell_id,
                                       d_cell_id,
                                       10*sizeof(int),
                                       hipMemcpyDeviceToHost));

            THEN("Then the global cell_id = {15, 8, 15, 14, 24, 5, 24, 24, 3, 15} ") {
                REQUIRE(test_cell_id[0] == 15);
                REQUIRE(test_cell_id[1] == 8);
                REQUIRE(test_cell_id[2] == 15);
                REQUIRE(test_cell_id[3] == 14);
                REQUIRE(test_cell_id[4] == 24);
                REQUIRE(test_cell_id[5] == 5);
                REQUIRE(test_cell_id[6] == 24);
                REQUIRE(test_cell_id[7] == 24);
                REQUIRE(test_cell_id[8] == 3);
                REQUIRE(test_cell_id[9] == 15);
            }

            hipFree(d_cell_id);
        }
        checkCudaErrors(hipblasDestroy(cublas_handle));
        hipFree(d_pos);
    }
}

SCENARIO("[DEVICE] Sort atoms", "[d-sort]") {
    GIVEN("An array of 10 known indices, with an associated atom_id array") {
        int atom_id[10] = {0,  1, 2,  3,  4,  5, 6,  7,  8, 9};
        int cell_id[10] = {15, 8, 15, 14, 24, 5, 24, 24, 3, 15};

        int *d_atom_id;
        int *d_cell_id;

        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_atom_id),
                                   10*sizeof(int)));
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_cell_id),
                                   10*sizeof(int)));

        checkCudaErrors(hipMemcpy(d_atom_id,
                                   atom_id,
                                   10*sizeof(int),
                                   hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_cell_id,
                                   cell_id,
                                   10*sizeof(int),
                                   hipMemcpyHostToDevice));


        WHEN("The sort_atoms function is called") {
            sort_atoms(10,
                       d_cell_id,
                       d_atom_id);

            checkCudaErrors(hipMemcpy(atom_id,
                                       d_atom_id,
                                       10*sizeof(int),
                                       hipMemcpyDeviceToHost));

            THEN("Then the global atom_id = {8, 5, 1, 3, 0, 2, 9, 4, 6, 7} ") {
                REQUIRE(atom_id[0] == 8);
                REQUIRE(atom_id[1] == 5);
                REQUIRE(atom_id[2] == 1);
                REQUIRE(atom_id[3] == 3);
                REQUIRE(atom_id[4] == 0);
                REQUIRE(atom_id[5] == 2);
                REQUIRE(atom_id[6] == 9);
                REQUIRE(atom_id[7] == 4);
                REQUIRE(atom_id[8] == 6);
                REQUIRE(atom_id[9] == 7);
            }
        }

        hipFree(d_atom_id);
        hipFree(d_cell_id);
    }
}

SCENARIO("[DEVICE] Count atoms", "[d-count]") {
    GIVEN("An array of 10 sorted cell_ids with num_cells = 8.") {
        int num_atoms = 10;
        int num_cells = 8;

        int cell_id[10] = {0, 2, 4, 5, 6, 6, 6, 8, 8, 8};
        int *d_cell_id;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_cell_id),
                                   num_atoms*sizeof(int)));
        checkCudaErrors(hipMemcpy(d_cell_id,
                                   cell_id,
                                   num_atoms*sizeof(int),
                                   hipMemcpyHostToDevice));

        WHEN("The sort_atoms function is called") {
            int *d_cell_num_atoms;
            int *d_cell_cumulative_num_atoms;

            int2 *d_cell_start_end;
            
            checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_cell_num_atoms),
                                       (num_cells+1)*sizeof(int)));
            checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_cell_cumulative_num_atoms),
                                       (num_cells+1)*sizeof(int)));
            checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_cell_start_end),
                                       (num_cells+1)*sizeof(int2)));

            // checkCudaErrors(hipMemset(d_cell_start_end,
            //                            -1,
            //                            num_atoms*sizeof(int2)));
            negative_elements<<<num_cells+1,1>>>(num_cells+1,
                                                 d_cell_start_end);

            count_atoms(num_atoms,
                        num_cells,
                        d_cell_id,
                        d_cell_start_end,
                        d_cell_num_atoms,
                        d_cell_cumulative_num_atoms);

            int t_cell_num_atoms[9];
            int t_cell_cumulative_num_atoms[9];
            int2 t_cell_start_end[9];

            checkCudaErrors(hipMemcpy(t_cell_num_atoms,
                                       d_cell_num_atoms,
                                       (num_cells+1)*sizeof(int),
                                       hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(t_cell_cumulative_num_atoms,
                                       d_cell_cumulative_num_atoms,
                                       (num_cells+1)*sizeof(int),
                                       hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(t_cell_start_end,
                                       d_cell_start_end,
                                       (num_cells+1)*sizeof(int2),
                                       hipMemcpyDeviceToHost));

            hipFree(d_cell_num_atoms);
            hipFree(d_cell_cumulative_num_atoms);
            hipFree(d_cell_start_end);

            THEN("Then the global cell_start_end = {{0, 0}, {-1, -1}, {1, 1}, {-1, -1}, {2, 2}, {3, 3}, {4, 6}, {7, 9}}") {
                REQUIRE(t_cell_start_end[0] == make_int2(0, 0));
                REQUIRE(t_cell_start_end[1] == make_int2(-1, -1));
                REQUIRE(t_cell_start_end[2] == make_int2(1, 1));
                REQUIRE(t_cell_start_end[3] == make_int2(-1, -1));
                REQUIRE(t_cell_start_end[4] == make_int2(2, 2));
                REQUIRE(t_cell_start_end[5] == make_int2(3, 3));
                REQUIRE(t_cell_start_end[6] == make_int2(4, 6));
                REQUIRE(t_cell_start_end[7] == make_int2(-1, -1));
                REQUIRE(t_cell_start_end[8] == make_int2(7, 9));
            }

            THEN("Then the global cell_num_atoms = {1, 0, 1, 0, 1, 1, 3, 0, 3}") {
                REQUIRE(t_cell_num_atoms[0] == 1);
                REQUIRE(t_cell_num_atoms[1] == 0);
                REQUIRE(t_cell_num_atoms[2] == 1);
                REQUIRE(t_cell_num_atoms[3] == 0);
                REQUIRE(t_cell_num_atoms[4] == 1);
                REQUIRE(t_cell_num_atoms[5] == 1);
                REQUIRE(t_cell_num_atoms[6] == 3);
                REQUIRE(t_cell_num_atoms[7] == 0);
                REQUIRE(t_cell_num_atoms[8] == 3);
            }

            THEN("Then the global cell_cumulative_num_atoms = {0, 1, 1, 2, 2, 3, 4, 7, 7}") {
                REQUIRE(t_cell_cumulative_num_atoms[0] == 0);
                REQUIRE(t_cell_cumulative_num_atoms[1] == 1);
                REQUIRE(t_cell_cumulative_num_atoms[2] == 1);
                REQUIRE(t_cell_cumulative_num_atoms[3] == 2);
                REQUIRE(t_cell_cumulative_num_atoms[4] == 2);
                REQUIRE(t_cell_cumulative_num_atoms[5] == 3);
                REQUIRE(t_cell_cumulative_num_atoms[6] == 4);
                REQUIRE(t_cell_cumulative_num_atoms[7] == 7);
                REQUIRE(t_cell_cumulative_num_atoms[8] == 7);
            }
        }

        hipFree(d_cell_id);
    }
}

SCENARIO("[DEVICE] Collide atoms", "[d-collide]") {
    GIVEN("An array of 10 atoms in a single cell.") {
        int num_atoms = 10;
        FN = 100;
        int num_cells = 1;
        double dt = 100*1.e-6;

        double3 vel[num_atoms];
        // Nothing special about these velcoities
        // They are just randomly generated for T=20uK
        vel[0] = make_double3( 0.034,-0.079, 0.006);
        vel[1] = make_double3(-0.012, 0.025,-0.012);
        vel[2] = make_double3(-0.044, 0.018,-0.031);
        vel[3] = make_double3( 0.064,-0.025,-0.009);
        vel[4] = make_double3(-0.006,-0.017, 0.017);
        vel[5] = make_double3(-0.000,-0.023, 0.052);
        vel[6] = make_double3( 0.063, 0.018, 0.069);
        vel[7] = make_double3( 0.021, 0.022, 0.002);
        vel[8] = make_double3(-0.032,-0.127,-0.074);
        vel[9] = make_double3( 0.066, 0.022, 0.075);
        double3 *d_vel;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_vel),
                                   num_atoms*sizeof(double3)));
        checkCudaErrors(hipMemcpy(d_vel,
                                   vel,
                                   num_atoms*sizeof(double3),
                                   hipMemcpyHostToDevice));

        int cell_id[10] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
        int *d_cell_id;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_cell_id),
                                   num_atoms*sizeof(int)));
        checkCudaErrors(hipMemcpy(d_cell_id,
                                   cell_id,
                                   num_atoms*sizeof(int),
                                   hipMemcpyHostToDevice));

        int cell_cumulative_num_atoms[2] = {0, 10};
        int *d_cell_cumulative_num_atoms;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_cell_cumulative_num_atoms),
                                   (num_cells+1)*sizeof(int)));
        checkCudaErrors(hipMemcpy(d_cell_cumulative_num_atoms,
                                   cell_cumulative_num_atoms,
                                   (num_cells+1)*sizeof(int),
                                   hipMemcpyHostToDevice));

        hiprandState *state;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&state),
                                   num_cells*sizeof(hiprandState)));
        initialise_rng_states(num_cells,
                              state);

        int *d_collision_count;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_collision_count),
                                   num_cells*sizeof(int)));
        checkCudaErrors(hipMemset(d_collision_count,
                                   0,
                                   num_cells*sizeof(int)));

        double *d_collision_remainder;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_collision_remainder),
                                   total_num_cells*sizeof(double)));

        double sig_vr_max[1] = {sqrt(16.*kB*20.e-6/h_pi/mass)*cross_section};
        double *d_sig_vr_max;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_sig_vr_max),
                                   num_cells*sizeof(double)));
        checkCudaErrors(hipMemcpy(d_sig_vr_max,
                                   sig_vr_max,
                                   num_cells*sizeof(double),
                                   hipMemcpyHostToDevice));

        // Make cell really small so that we can have collisions between the ten atoms
        copy_collision_params_to_device<<<1, 1>>>(make_double3(0., 0., 0.),
                                                  make_double3(2.5e-6, 2.5e-6, 2.5e-6),
                                                  make_int3(1,1,1),
                                                  FN);

        WHEN("The cu_collide function is called once") {

            cu_collide(num_cells,
                       d_cell_id,
                       d_cell_cumulative_num_atoms,
                       dt,
                       state,
                       d_collision_count,
                       d_collision_remainder,
                       d_sig_vr_max,
                       d_vel);

            int t_collision_count[num_cells];
            double t_sig_vr_max[num_cells];

            checkCudaErrors(hipMemcpy(t_collision_count,
                                       d_collision_count,
                                       num_cells*sizeof(int),
                                       hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(t_sig_vr_max,
                                       d_sig_vr_max,
                                       num_cells*sizeof(double),
                                       hipMemcpyDeviceToHost));

            THEN("We should expect zero simulated collisions") {
                REQUIRE(t_collision_count[0] == 0*FN);
            }
            THEN("The sig_vr_max array should not have been updated") {
                REQUIRE(t_sig_vr_max[0] == sig_vr_max[0]);
            }
        }

        hipFree(d_vel);
        hipFree(d_cell_id);
        hipFree(d_cell_cumulative_num_atoms);
        hipFree(state);
        hipFree(d_collision_count);
        hipFree(d_sig_vr_max);
    }
}

SCENARIO("[DEVICE] Collision rate", "[d-collrate]") {
    GIVEN("An array of 10,000 thermal atoms.") {
        int num_atoms = 1e5;
        FN = 10;
        
        // Initialise grid parameters
        k_num_cells = make_int3(35, 35, 35);
        total_num_cells = k_num_cells.x*k_num_cells.y*k_num_cells.z;
        
        double dt = 100*1.e-6;

        hiprandState *state;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&state),
                                   num_atoms*sizeof(hiprandState)));
        initialise_rng_states(num_atoms,
                              state);

        double3 *d_vel;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_vel),
                                   num_atoms*sizeof(double3)));
        // Generate velocity distribution
        generate_thermal_velocities(num_atoms,
                                    20.e-6,
                                    state,
                                    d_vel);

#if defined(IP)  // Ioffe Pritchard trap
        trap_geo trap_parameters;
        trap_parameters.B0 = 0.01;
        trap_parameters.dB = 20.;
        trap_parameters.ddB = 40000.;
#else  // Quadrupole trap
        trap_geo trap_parameters;
        trap_parameters.Bz = 2.0;
        trap_parameters.B0 = 0.;
#endif

        double3 *d_pos;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_pos),
                                   num_atoms*sizeof(double3)));
        // Generate position distribution
        generate_thermal_positions(num_atoms,
                                   20.e-6,
                                   trap_parameters,
                                   state,
                                   d_pos);

        int *d_cell_id;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_cell_id),
                                   num_atoms*sizeof(int)));
        checkCudaErrors(hipMemset(d_cell_id,
                                   0,
                                   num_atoms*sizeof(int)));

        int *d_atom_id;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_atom_id),
                                   num_atoms*sizeof(int)));

        initialise_atom_id(num_atoms,
                           d_atom_id);

        int2 *d_cell_start_end;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_cell_start_end),
                                   (total_num_cells+1)*sizeof(int2)));
        // checkCudaErrors(hipMemset(d_cell_start_end,
        //                            -1,
        //                            (total_num_cells+1)*sizeof(int2)));
        negative_elements<<<total_num_cells+1,1>>>(total_num_cells+1,
                                                 d_cell_start_end);

        int *d_cell_num_atoms;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_cell_num_atoms),
                                   (total_num_cells+1)*sizeof(int)));
        checkCudaErrors(hipMemset(d_cell_num_atoms,
                                   0,
                                   (total_num_cells+1)*sizeof(int)));

        int *d_cell_cumulative_num_atoms;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_cell_cumulative_num_atoms),
                                   (total_num_cells+1)*sizeof(int)));
        checkCudaErrors(hipMemset(d_cell_cumulative_num_atoms,
                                   0,
                                   (total_num_cells+1)*sizeof(int)));

        int *d_collision_count;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_collision_count),
                                   total_num_cells*sizeof(int)));
        checkCudaErrors(hipMemset(d_collision_count,
                                   0,
                                   total_num_cells*sizeof(int)));
        double *d_collision_remainder;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_collision_remainder),
                                   total_num_cells*sizeof(double)));
        // checkCudaErrors(hipMemset(d_collision_remainder,
        //                            0.,
        //                            total_num_cells*sizeof(double)));
        zero_elements<<<total_num_cells,1>>>(total_num_cells,
                                             d_collision_remainder);

        double sig_vr_max[total_num_cells];
        for (int cell = 0; cell < total_num_cells; ++cell) {
             sig_vr_max[cell] = sqrt(16.*kB*20.e-6/h_pi/mass)*cross_section;
        }
        double *d_sig_vr_max;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_sig_vr_max),
                                   total_num_cells*sizeof(double)));
        checkCudaErrors(hipMemcpy(d_sig_vr_max,
                                   sig_vr_max,
                                   total_num_cells*sizeof(double),
                                   hipMemcpyHostToDevice));

        hipblasHandle_t cublas_handle;
        checkCudaErrors(hipblasCreate(&cublas_handle));
        // Set up global grid parameters
        initialise_grid_params(num_atoms,
                               cublas_handle,
                               d_pos);

        WHEN("The collide_atoms function is called one hundred times") {
            for (int i = 0; i < 100; ++i) {
                collide_atoms(num_atoms,
                              total_num_cells,
                              dt,
                              d_pos,
                              d_vel,
                              state,
                              d_sig_vr_max,
                              d_cell_id,
                              d_atom_id,
                              d_cell_start_end,
                              d_cell_num_atoms,
                              d_cell_cumulative_num_atoms,
                              d_collision_remainder,
                              d_collision_count);
                progress_bar(i,
                             100);
            }

            int t_collision_count[total_num_cells];
            checkCudaErrors(hipMemcpy(t_collision_count,
                                       d_collision_count,
                                       total_num_cells*sizeof(int),
                                       hipMemcpyDeviceToHost));

            int total_coll = 0;
            for (int cell = 0; cell < total_num_cells; ++cell) {
                total_coll += t_collision_count[cell];
            }

#if defined(IP)  // Ioffe Pritchard trap
            THEN("We should expect the collision rate to agree with Walraven") {
                REQUIRE(total_coll < 2407 * (1+fractional_tol));
                REQUIRE(total_coll > 2407 * (1-fractional_tol));
            }
#else  // Quadrupole
            THEN("We should expect the collision rate to agree with Walraven") {
                REQUIRE(total_coll < 1026 * (1+fractional_tol));
                REQUIRE(total_coll > 1026 * (1-fractional_tol));
            }
#endif
        }

        checkCudaErrors(hipblasDestroy(cublas_handle));

        hipFree(d_pos);
        hipFree(d_vel);
        hipFree(state);
        hipFree(d_sig_vr_max);
        hipFree(d_cell_id);
        hipFree(d_atom_id);
        hipFree(d_cell_start_end);
        hipFree(d_cell_num_atoms);
        hipFree(d_cell_cumulative_num_atoms);
        hipFree(d_collision_count);
        hipFree(d_collision_remainder);
    }
}

__global__ void copy_d_grid_min(double3 *grid_min) {
    grid_min[0] = d_grid_min;
    return;
}

__global__ void copy_d_cell_length(double3 *cell_length) {
    cell_length[0] = d_cell_length;
    return;
}
