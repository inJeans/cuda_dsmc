#include "hip/hip_runtime.h"
/** \file
 *  \brief Unit tests for the distribution_generation file
 *
 *  More detailed description
 *  Copyright 2015 Christopher Watkins
 */

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <float.h>
#include <algorithm>

#define CATCH_CONFIG_MAIN
#include "catch.hpp"
extern "C"
{
#include "unif01.h"
#include "bbattery.h" 
}

#include "random_numbers.hpp"
#include "hip/hip_runtime_api.h"
#include "random_number_generation_tests.cuh"

#include "define_host_constants.hpp"

SCENARIO("[DEVICE] Uniform random number generation", "[d-urng]") {
    GIVEN("An appropriate seed") {
        hiprandState *state;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&state),
                                   sizeof(hiprandState)));
        initialise_rng_states(1,
                              state);

        WHEN("The random number generator is called") {
            double r;
            uniform_prng(1,
                         state,
                         &r);

            THEN("The result should be between 0 and 1") {
                REQUIRE(r >= 0.);
                REQUIRE(r <= 1.);
            }
        }

        // WHEN("We assign the local seed to the global seed") {
        //     g_rng = rng;
        //     unif01_Gen *gen;
        //     char* rng_name = "g_uniform_prng";
        //     gen = unif01_CreateExternGen01(rng_name,
        //                                    g_uniform_prng);

        //     THEN("We expect to pass small crush") {
        //         bbattery_SmallCrush(gen);
        //         bool complete = true;
        //         REQUIRE(complete);
        //     }

        //     unif01_DeleteExternGen01(gen);
        // }

        hipFree(state);
    }
}

SCENARIO("[DEVICE] Normally distributed random number generation", "[d-nrng]") {
    GIVEN("An array of appropriate seeds") {
        int num_test = 10000;

        hiprandState *state;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&state),
                                   num_test*sizeof(hiprandState)));
        initialise_rng_states(num_test,
                              state);

        WHEN("We generate 10,000 numbers using a mean of 0 and a std of 1") {
            double *d_test_values;
            checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_test_values),
                                       num_test*sizeof(double)));
            gaussian_prng(num_test,
                          state,
                          d_test_values);

            double *test_values;
            test_values = reinterpret_cast<double*>(calloc(num_test,
                                                    sizeof(double)));
            checkCudaErrors(hipMemcpy(test_values,
                                       d_test_values,
                                       num_test*sizeof(double),
                                       hipMemcpyDeviceToHost));

            THEN("The result should pass the back-of-the-envelope test") {
                double val_mean = mean(test_values,
                                       num_test);
                double val_std  = std_dev(test_values,
                                          num_test);
                double val_max = *std::max_element(test_values,
                                                   test_values+num_test);
                double val_min = *std::min_element(test_values,
                                                   test_values+num_test);

                double Z_max = z_score(val_max,
                                       val_mean,
                                       val_std);
                double Z_min = z_score(val_min,
                                       val_mean,
                                       val_std);
                REQUIRE(Z_max <= 4.);
                REQUIRE(Z_min >=-4.);
            }

            hipFree(d_test_values);
            free(test_values);
            // Also need to implement a more rigorous test
        }

        hipFree(state);
    }
}

__host__ void uniform_prng(int num_elements,
                           hiprandState *state,
                           double *h_r) {
    double *d_r;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_r),
                               num_elements*sizeof(double)));

    g_uniform_prng<<<1,1>>>(num_elements,
                            state,
                            d_r);

    checkCudaErrors(hipMemcpy(h_r,
                               d_r,
                               num_elements*sizeof(double),
                               hipMemcpyDeviceToHost));

    return;
}

__global__ void g_uniform_prng(int num_elements,
                               hiprandState *state,
                               double *r) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < num_elements;
         i += blockDim.x * gridDim.x) {
        r[i] = hiprand_uniform(state);
    }

    return;
}

__host__ void gaussian_prng(int num_elements,
                           hiprandState *state,
                           double *h_r) {
    double *d_r;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_r),
                               num_elements*sizeof(double)));

    g_gaussian_prng<<<num_elements,1>>>(num_elements,
                                        state,
                                        d_r);

    checkCudaErrors(hipMemcpy(h_r,
                               d_r,
                               num_elements*sizeof(double),
                               hipMemcpyDeviceToHost));

    return;
}

__global__ void g_gaussian_prng(int num_elements,
                                hiprandState *state,
                                double *r) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < num_elements;
         i += blockDim.x * gridDim.x) {
        r[i] = hiprand_normal(state);
    }

    return;
}

double mean(double *array,
            int num_elements) {
    double mean = 0.;
    for (int i = 0; i < num_elements; ++i)
        mean += array[i];

    return mean / num_elements;
}

double std_dev(double *array,
               int num_elements) {
    double mu = mean(array,
                     num_elements);
    double sum = 0.;
    for (int i = 0; i < num_elements; ++i)
        sum += (array[i]-mu) * (array[i]-mu);

    return sqrt(sum / num_elements);
}

double z_score(double value,
               double mean,
               double std) {
    return (value - mean) / std;
}
