#include "hip/hip_runtime.h"
/** \file
 *  \brief Unit tests for the collisions file
 *
 *  More detailed description
 *  Copyright 2015 Christopher Watkins
 */

#include "ehrenfest_test.cuh"
#if defined(OMP)
    #include <omp.h>
#endif

double fractional_tol = 0.05; 

SCENARIO("[DEVICE] Execute a full ehrenfest simulation", "[d-ehrenfest]") {
    GIVEN("A thermally distributed intial distribution.") {

        // Initialise trapping parameters
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the trapping parameters.");
#endif
#if defined(IP)  // Ioffe Pritchard trap
        trap_geo trap_parameters;
        trap_parameters.B0 = 0.01;
        trap_parameters.dB = 20.;
        trap_parameters.ddB = 40000.;
#else  // Quadrupole trap
        trap_geo trap_parameters;
        trap_parameters.Bz = 2.0;
        trap_parameters.B0 = 0.;
#endif

        int devID = gpuGetMaxGflopsDeviceId();
        checkCudaErrors(hipSetDevice(devID));

        int device_count;
        hipGetDeviceCount(&device_count);

        int num_batches = 1;
#if defined(OMP)
        num_batches = omp_get_max_threads();
#else
        num_batches = device_count;
#endif

        // Initialise computational parameters.
        int num_atoms = 1e5;
        FN = 10;
        
        double dt = 1.e-7;
        int num_time_steps = 50;
        int loops_per_collision = 10000;
        double init_temp = 20.e-6;

        int b_num_atoms[num_batches];
        for (int batch=0; batch < num_batches; ++batch) {
            b_num_atoms[batch] = num_atoms / num_batches;
        }
        b_num_atoms[num_batches-1] = num_atoms - (num_batches-1)*(num_atoms/num_batches);

        // Initialise grid parameters
        k_num_cells = make_int3(35, 35, 35);
        total_num_cells = k_num_cells.x*k_num_cells.y*k_num_cells.z;

        int largest = 0;
        if (num_atoms > total_num_cells) {
            largest = num_atoms;
        } else {
            largest = total_num_cells;
        }

    // Initialise rng
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the rng state array.");
        LOGF(DEBUG, "\nAllocating %i hiprandState elements on the device.",
             largest);
#endif
        hiprandState *state;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&state),
                                   largest*sizeof(hiprandState)));
        initialise_rng_states(largest,
                              state,
                              false);

        // Initialise atom_id
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the atom_id array.");
        LOGF(DEBUG, "\nAllocating %i int elements on the device.",
             num_atoms);
#endif
        int *atom_id;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&atom_id),
                                   num_atoms*sizeof(int)));
        initialise_atom_id(num_atoms,
                           atom_id);

        // Initialise cell_id
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the cell_id array.");
        LOGF(DEBUG, "\nAllocating %i int elements on the device.",
             num_atoms);
#endif
        int *cell_id;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&cell_id),
                                   num_atoms*sizeof(int)));
        checkCudaErrors(hipMemset(cell_id,
                                   0,
                                   num_atoms*sizeof(int)));

        // Initialise cell_start_end
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the cell_start_end array.");
        LOGF(DEBUG, "\nAllocating %i int2 elements on the device.",
             total_num_cells+1);
#endif
        int2 *cell_start_end;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&cell_start_end),
                                   (total_num_cells+1)*sizeof(int2)));
        checkCudaErrors(hipMemset(cell_start_end,
                                   -1,
                                   (total_num_cells+1)*sizeof(int2)));

        // Initialise cell_num_atoms
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the cell_num_atoms array.");
        LOGF(DEBUG, "\nAllocating %i int elements on the device.",
             total_num_cells+1);
#endif
        int *cell_num_atoms;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&cell_num_atoms),
                                   (total_num_cells+1)*sizeof(int)));
        checkCudaErrors(hipMemset(cell_num_atoms,
                                   0,
                                   (total_num_cells+1)*sizeof(int)));

        // Initialise cell_cumulative_num_atoms
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the cell_cumulative_num_atoms array.");
        LOGF(DEBUG, "\nAllocating %i int elements on the device.",
             total_num_cells+1);
#endif
        int *cell_cumulative_num_atoms;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&cell_cumulative_num_atoms),
                                   (total_num_cells+1)*sizeof(int)));
        checkCudaErrors(hipMemset(cell_cumulative_num_atoms,
                                   0,
                                   (total_num_cells+1)*sizeof(int)));

        // Initialise collision_count
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the collision_count array.");
        LOGF(DEBUG, "\nAllocating %i int elements on the device.",
             total_num_cells);
#endif
        double *collision_count;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&collision_count),
                                   (total_num_cells)*sizeof(double)));
        checkCudaErrors(hipMemset(collision_count,
                                   0.,
                                   (total_num_cells)*sizeof(double)));

        // Initialise collision_remainder
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the collision_remainder array.");
        LOGF(DEBUG, "\nAllocating %i int elements on the device.",
             total_num_cells);
#endif
        double *collision_remainder;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&collision_remainder),
                                   (total_num_cells)*sizeof(double)));
        zero_elements<<<total_num_cells,1>>>(total_num_cells,
                                             collision_remainder);

        // Initialise sig_vr_max
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the sig_vr_max array.");
        LOGF(DEBUG, "\nAllocating %i int elements on the device.",
             total_num_cells);
#endif
        double h_sig_vr_max[total_num_cells];
        for (int cell = 0; cell < total_num_cells; ++cell) {
             h_sig_vr_max[cell] = sqrt(16.*kB*20.e-6/h_pi/mass)*cross_section;
        }
        double *sig_vr_max;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&sig_vr_max),
                                   total_num_cells*sizeof(double)));
        checkCudaErrors(hipMemcpy(sig_vr_max,
                                   h_sig_vr_max,
                                   total_num_cells*sizeof(double),
                                   hipMemcpyHostToDevice));

    // Initialise velocities
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the velocity array.");
        LOGF(DEBUG, "\nAllocating %i double3 elements on the device.",
             num_atoms);
#endif
        double3 *vel;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&vel),
                                   num_atoms*sizeof(double3)));

        // Generate velocity distribution
        generate_thermal_velocities(num_atoms,
                                    init_temp,
                                    state,
                                    vel);

        // Initialise positions
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the position array.");
        LOGF(DEBUG, "\nAllocating %i double3 elements on the device.",
             num_atoms);
#endif
        double3 *pos;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&pos),
                                   num_atoms*sizeof(double3)));

        // Generate position distribution
        generate_thermal_positions(num_atoms,
                                   init_temp,
                                   trap_parameters,
                                   state,
                                   pos);

        // Initialise wavefunction
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the wavefunction array.");
        LOGF(DEBUG, "\nAllocating %i wavefunction elements on the device.",
             num_atoms);
#endif  // Logging
        wavefunction *psi;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&psi),
                                   num_atoms*sizeof(wavefunction)));

        // Generate wavefunction
        generate_aligned_spins(num_atoms,
                               trap_parameters,
                               pos,
                               psi);

        // Initialise accelerations
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the acceleration array.");
        LOGF(DEBUG, "\nAllocating %i double3 elements on the device.",
             num_atoms);
#endif
        double3 *acc;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&acc),
                                   num_atoms*sizeof(double3)));

        // Generate accelerations
        update_accelerations(num_atoms,
                             trap_parameters,
                             pos,
                             acc,
                             psi);

        // Initialise kinetic energy
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the kinetic energy array.");
        LOGF(DEBUG, "\nAllocating %i double elements on the device.",
             num_atoms);
#endif
        double *d_kinetic_energy;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_kinetic_energy),
                                   num_atoms*sizeof(double)));
        double *avg_kinetic_energy;
        avg_kinetic_energy = reinterpret_cast<double*>(calloc(num_time_steps+1,
                                                              sizeof(double)));

        // Initialise potential energy
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the potential energy array.");
        LOGF(DEBUG, "\nAllocating %i double elements on the device.",
             num_atoms);
#endif
        double *d_potential_energy;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_potential_energy),
                                   num_atoms*sizeof(double)));
        double *avg_potential_energy;
        avg_potential_energy = reinterpret_cast<double*>(calloc(num_time_steps+1,
                                                                sizeof(double)));

        // Initialise projection
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the projection array.");
        LOGF(DEBUG, "\nAllocating %i double elements on the device.",
             num_atoms);
#endif
        double *d_projection;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_projection),
                                   num_atoms*sizeof(double)));
        double *avg_projection;
        avg_projection = reinterpret_cast<double*>(calloc(num_time_steps+1,
                                                          sizeof(double)));

        // Initialise spin up
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the spin up array.");
        LOGF(DEBUG, "\nAllocating %i int elements on the device.",
             num_atoms);
#endif
        int *is_spin_up;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&is_spin_up),
                                   num_atoms*sizeof(int)));
        int *num_spin_up;
        num_spin_up = reinterpret_cast<int*>(calloc(num_time_steps+1,
                                                    sizeof(int)));

        // initialise time
        double *sim_time;
        sim_time = reinterpret_cast<double*>(calloc(num_time_steps+1,
                                                    sizeof(double)));

#if defined(LOGGING)
        LOGF(DEBUG, "\nCreating the cuBLAS handle.\n");
#endif
        hipblasHandle_t cublas_handle;
        checkCudaErrors(hipblasCreate(&cublas_handle));

        // Set up global grid parameters
        initialise_grid_params(num_atoms,
                               cublas_handle,
                               pos);

        double3 *h_pos;
        h_pos = reinterpret_cast<double3*>(calloc(num_atoms,
                                                  sizeof(double3)));
        checkCudaErrors(hipMemcpy(h_pos,
                                   pos,
                                   num_atoms*sizeof(double3),
                                   hipMemcpyDeviceToHost));
        FILE *init_pos_file_pointer = fopen("initial_position.data", "w");
        for (int i=0; i<num_atoms; ++i) {
            fprintf(init_pos_file_pointer, "%g, %g, %g\n", h_pos[i].x,
                                                           h_pos[i].y,
                                                           h_pos[i].z);
        }
        fclose(init_pos_file_pointer);

        double *h_collision_count;
        h_collision_count = reinterpret_cast<double*>(calloc(total_num_cells,
                                                          sizeof(double)));

        FILE *collision_file_pointer = fopen("collision.data", "w");
        for (int i=0; i<total_num_cells; ++i) {
            fprintf(collision_file_pointer, "%g\t", h_collision_count[i]);
        }
        fprintf(collision_file_pointer, "\n");
        fclose(collision_file_pointer);

        sim_time[0] = 0.;

        avg_projection[0] = inst_projection(num_atoms,
                                            pos,
                                            trap_parameters,
                                            psi,
                                            d_projection) / num_atoms;

        num_spin_up[0] = inst_is_spin_up(num_atoms,
                                         psi,
                                         is_spin_up);

        avg_kinetic_energy[0] = inst_kinetic_energy(num_atoms,
                                                    vel,
                                                    psi,
                                                    d_kinetic_energy) /
                                 num_atoms;
        avg_potential_energy[0] = inst_potential_energy(num_atoms,
                                                        pos,
                                                        trap_parameters,
                                                        psi,
                                                        d_potential_energy) /
                                  num_atoms;

        double3 *b_pos[num_batches];
        double3 *b_vel[num_batches];
        double3 *b_acc[num_batches];
        wavefunction *b_psi[num_batches];
        int *b_cell_id[num_batches];
        hipblasHandle_t b_cublas_handle[num_batches];
        for (int batch = 0; batch < num_batches; ++batch) {
            checkCudaErrors(hipSetDevice(batch % device_count));
            checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&b_pos[batch]),
                                       b_num_atoms[batch]*sizeof(double3)));
            checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&b_vel[batch]),
                                       b_num_atoms[batch]*sizeof(double3)));
            checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&b_acc[batch]),
                                       b_num_atoms[batch]*sizeof(double3)));
            checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&b_psi[batch]),
                                       b_num_atoms[batch]*sizeof(wavefunction)));
            checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&b_cell_id[batch]),
                                       b_num_atoms[batch]*sizeof(int)));
            checkCudaErrors(hipblasCreate(&b_cublas_handle[batch]));
        }
        devID = gpuGetMaxGflopsDeviceId();
        checkCudaErrors(hipSetDevice(devID));

        // Evolve many time step
#if defined(LOGGING)
        LOGF(INFO, "\nEvolving distribution for %i time steps.", num_time_steps);
#endif
        for (int t = 0; t < num_time_steps; ++t) {
            #pragma omp parallel for
            for(int batch=0; batch < num_batches; ++batch) {
                checkCudaErrors(hipMemcpy(b_pos[batch],
                                           &pos[batch*num_atoms/num_batches],
                                           b_num_atoms[batch]*sizeof(double3),
                                           hipMemcpyDeviceToDevice));
                checkCudaErrors(hipMemcpy(b_vel[batch],
                                           &vel[batch*num_atoms/num_batches],
                                           b_num_atoms[batch]*sizeof(double3),
                                           hipMemcpyDeviceToDevice));
                checkCudaErrors(hipMemcpy(b_acc[batch],
                                           &acc[batch*num_atoms/num_batches],
                                           b_num_atoms[batch]*sizeof(double3),
                                           hipMemcpyDeviceToDevice));
                checkCudaErrors(hipMemcpy(b_psi[batch],
                                           &psi[batch*num_atoms/num_batches],
                                           b_num_atoms[batch]*sizeof(wavefunction),
                                           hipMemcpyDeviceToDevice));
            }
            for(int u=0; u < loops_per_collision; ++u) {
                #pragma omp parallel for
                for(int batch=0; batch < num_batches; ++batch) {
                    checkCudaErrors(hipSetDevice(batch % device_count));
                    velocity_verlet_update(b_num_atoms[batch],
                                           dt,
                                           trap_parameters,
                                           b_cublas_handle[batch],
                                           b_pos[batch],
                                           b_vel[batch],
                                           b_acc[batch],
                                           b_psi[batch]);
                }
            }

            // #pragma omp parallel for
            // for(int batch=0; batch < num_batches; ++batch) {
            //     checkCudaErrors(hipSetDevice(batch % device_count));
            //     // Index atoms
            //     index_atoms(b_num_atoms[batch],
            //                 b_pos[batch],
            //                 b_cell_id[batch]);
            // }
            devID = gpuGetMaxGflopsDeviceId();
            checkCudaErrors(hipSetDevice(devID));

            #pragma omp parallel for
            for(int batch=0; batch < num_batches; ++batch) {
                checkCudaErrors(hipMemcpy(&pos[batch*num_atoms/num_batches],
                                           b_pos[batch],
                                           b_num_atoms[batch]*sizeof(double3),
                                           hipMemcpyDeviceToDevice));
                checkCudaErrors(hipMemcpy(&vel[batch*num_atoms/num_batches],
                                           b_vel[batch],
                                           b_num_atoms[batch]*sizeof(double3),
                                           hipMemcpyDeviceToDevice));
                // checkCudaErrors(hipMemcpy(&acc[batch*num_atoms/num_batches],
                //                            b_acc[batch],
                //                            b_num_atoms[batch]*sizeof(double3),
                //                            hipMemcpyDeviceToDevice));
                // checkCudaErrors(hipMemcpy(&psi[batch*num_atoms/num_batches],
                //                            b_psi[batch],
                //                            b_num_atoms[batch]*sizeof(wavefunction),
                //                            hipMemcpyDeviceToDevice));
                checkCudaErrors(hipMemcpy(&cell_id[batch*num_atoms/num_batches],
                                           b_cell_id[batch],
                                           b_num_atoms[batch]*sizeof(int),
                                           hipMemcpyDeviceToDevice));
            }
            sim_time[t+1] = sim_time[t] + loops_per_collision*dt;
            checkCudaErrors(hipMemset(cell_start_end,
                                       -1,
                                       (total_num_cells+1)*sizeof(int2)));
            collide_atoms(num_atoms,
                          total_num_cells,
                          loops_per_collision*dt,
                          pos,
                          vel,
                          state,
                          sig_vr_max,
                          cell_id,
                          atom_id,
                          cell_start_end,
                          cell_num_atoms,
                          cell_cumulative_num_atoms,
                          collision_remainder,
                          collision_count);
            
            checkCudaErrors(hipMemcpy(h_collision_count,
                                       collision_count,
                                       total_num_cells*sizeof(double),
                                       hipMemcpyDeviceToHost));
            collision_file_pointer = fopen("collision.data", "a");
            for (int cell=0; cell < total_num_cells; ++cell) {
                fprintf(collision_file_pointer, "%g\t", h_collision_count[cell]);
            }
            fprintf(collision_file_pointer, "\n");
            fclose(collision_file_pointer);

            avg_projection[t+1] = inst_projection(num_atoms,
                                                  pos,
                                                  trap_parameters,
                                                  psi,
                                                  d_projection);
            num_spin_up[t+1] = inst_is_spin_up(num_atoms,
                                               psi,
                                               is_spin_up);
            avg_projection[t+1] /= num_spin_up[t+1];
            avg_kinetic_energy[t+1] = inst_kinetic_energy(num_atoms,
                                                          vel,
                                                          psi,
                                                          d_kinetic_energy) /
                                       num_spin_up[t+1];
            avg_potential_energy[t+1] = inst_potential_energy(num_atoms,
                                                              pos,
                                                              trap_parameters,
                                                              psi,
                                                              d_potential_energy) /
                                        num_spin_up[t+1];

            progress_bar(t,
                         num_time_steps-1);
        }

        FILE *time_file_pointer = fopen("time.data", "w");
        FILE *kinetic_file_pointer = fopen("kinetic_energy.data", "w");
        FILE *potential_file_pointer = fopen("potential_energy.data", "w");
        FILE *projection_file_pointer = fopen("projection.data", "w");
        FILE *spin_up_file_pointer = fopen("num_spin_up.data", "w");
        for (int i=0; i<num_time_steps+1; ++i) {
            fprintf(time_file_pointer, "%g\n", sim_time[i]);
            fprintf(kinetic_file_pointer, "%g\n", avg_kinetic_energy[i]/kB*1.e6);
            fprintf(potential_file_pointer, "%g\n", avg_potential_energy[i]/kB*1.e6);
            fprintf(projection_file_pointer, "%g\n", avg_projection[i]);
            fprintf(spin_up_file_pointer, "%i\n", num_spin_up[i]);
        }
        fclose(time_file_pointer);
        fclose(kinetic_file_pointer);
        fclose(potential_file_pointer);
        fclose(projection_file_pointer);
        fclose(spin_up_file_pointer);

        checkCudaErrors(hipMemcpy(h_pos,
                                   pos,
                                   num_atoms*sizeof(double3),
                                   hipMemcpyDeviceToHost));
        FILE *final_pos_file_pointer = fopen("final_position.data", "w");
        for (int i=0; i<num_atoms; ++i) {
            fprintf(final_pos_file_pointer, "%g, %g, %g\n", h_pos[i].x,
                                                            h_pos[i].y,
                                                            h_pos[i].z);
        }
        fclose(final_pos_file_pointer);

#if defined(IP)  // Ioffe Pritchard trap
            THEN("We should expect the collision rate to agree with Walraven") {
                // REQUIRE(total_coll < 2407 * (1+fractional_tol));
                // REQUIRE(total_coll > 2407 * (1-fractional_tol));
            }
#else  // Quadrupole
            THEN("We should expect the collision rate to agree with Walraven") {
                // REQUIRE(total_coll < 1026 * (1+fractional_tol));
                // REQUIRE(total_coll > 1026 * (1-fractional_tol));
            }
#endif

#if defined(LOGGING)
        LOGF(DEBUG, "\nDestroying the cuBLAS handle.\n");
#endif
        checkCudaErrors(hipblasDestroy(cublas_handle));

#if defined(LOGGING)
        LOGF(INFO, "\nCleaning up device memory.");
#endif
        hipFree(state);
        hipFree(atom_id);
        hipFree(cell_id);
        hipFree(cell_start_end);
        hipFree(cell_num_atoms);
        hipFree(cell_cumulative_num_atoms);
        hipFree(collision_count);
        hipFree(collision_remainder);
        hipFree(sig_vr_max);
        hipFree(vel);
        hipFree(pos);
        hipFree(acc);
        hipFree(psi);
        hipFree(d_kinetic_energy);
        hipFree(d_potential_energy);
        hipFree(d_projection);
        hipFree(is_spin_up);

        for (int batch = 0; batch < num_batches; ++batch) {
            checkCudaErrors(hipSetDevice(batch % device_count));
            hipFree(b_pos[batch]);
            hipFree(b_vel[batch]);
            hipFree(b_acc[batch]);
            hipFree(b_psi[batch]);
            hipFree(b_cell_id[batch]);
            checkCudaErrors(hipblasDestroy(b_cublas_handle[batch]));
        }

        free(avg_kinetic_energy);
        free(avg_potential_energy);
        free(avg_projection);
        free(num_spin_up);
        free(sim_time);
        free(h_pos);
        free(h_collision_count);
    }
}

__host__ double inst_kinetic_energy(int num_atoms,
                                    double3 *vel,
                                    wavefunction *psi,
                                    double *kinetic_energy) {
    double *h_inst_kin = NULL;
    h_inst_kin = reinterpret_cast<double*>(calloc(1,
                                                  sizeof(double)));
    double *d_inst_kin = NULL;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_inst_kin),
                               sizeof(double)));

    cu_kinetic_energy(num_atoms,
                      vel,
                      psi,
                      kinetic_energy);
    // Determine temporary device storage requirements
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage,
                                           temp_storage_bytes,
                                           kinetic_energy,
                                           d_inst_kin,
                                           num_atoms));
    // Allocate temporary storage
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_temp_storage),
                               temp_storage_bytes));
    // Run sum-reduction
    checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage,
                                           temp_storage_bytes,
                                           kinetic_energy,
                                           d_inst_kin,
                                           num_atoms));
    checkCudaErrors(hipMemcpy(h_inst_kin,
                               d_inst_kin,
                               1.*sizeof(double),
                               hipMemcpyDeviceToHost));
    hipFree(d_temp_storage);
    hipFree(d_inst_kin);

    return h_inst_kin[0];
}

__host__ void cu_kinetic_energy(int num_atoms,
                                double3 *vel,
                                wavefunction *psi,
                                double *kinetic_energy) {
#if defined(LOGGING)
    LOGF(DEBUG, "\nCalculating optimal launch configuration for the kinetic "
                "energy calculation kernel.\n");
#endif
    int block_size = 0;
    int min_grid_size = 0;
    int grid_size = 0;
    hipOccupancyMaxPotentialBlockSize(&min_grid_size,
                                       &block_size,
                                       (const void *) g_kinetic_energy,
                                       0,
                                       num_atoms);
    grid_size = (num_atoms + block_size - 1) / block_size;
#if defined(LOGGING)
    LOGF(DEBUG, "\nLaunch config set as <<<%i,%i>>>\n",
                grid_size, block_size);
#endif

    g_kinetic_energy<<<grid_size,
                       block_size>>>
                      (num_atoms,
                       vel,
                       psi,
                       kinetic_energy);

    return;
}

__global__ void g_kinetic_energy(int num_atoms,
                                 double3 *vel,
                                 wavefunction *psi,
                                 double *kinetic_energy) {
    for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
         atom < num_atoms;
         atom += blockDim.x * gridDim.x) {
        kinetic_energy[atom] = d_kinetic_energy(vel[atom],
                                                psi[atom]);
        if(kinetic_energy[atom] != kinetic_energy[atom]) {
            kinetic_energy[atom] = 0.;
            vel[atom] = make_double3(0., 0., 0.);
        }
    }

    return;
}

__device__ double d_kinetic_energy(double3 vel,
                                   wavefunction psi) {
    double kinetic = 0.;
    if (psi.isSpinUp) {
        kinetic = 0.5 * d_mass * norm(vel) * norm(vel);
    }
    return kinetic;
}

__host__ double inst_potential_energy(int num_atoms,
                                      double3 *pos,
                                      trap_geo params,
                                      wavefunction *psi,
                                      double *potential_energy) { 
    double *h_inst_pot = NULL;
    h_inst_pot = reinterpret_cast<double*>(calloc(1, 
                                                  sizeof(double)));
    double *d_inst_pot = NULL;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_inst_pot),
                               sizeof(double)));
    cu_potential_energy(num_atoms,
                        pos,
                        params,
                        psi,
                        potential_energy);
    // Determine temporary device storage requirements 
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0; 
    checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage,
                                           temp_storage_bytes,
                                           potential_energy,
                                           d_inst_pot,
                                           num_atoms));
    // Allocate temporary storage
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_temp_storage),
                               temp_storage_bytes));
    // Run sum-reduction
    checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage,
                                           temp_storage_bytes,
                                           potential_energy,
                                           d_inst_pot,
                                           num_atoms));
    checkCudaErrors(hipMemcpy(h_inst_pot,
                               d_inst_pot,
                               sizeof(double),
                               hipMemcpyDeviceToHost));
    hipFree(d_temp_storage);
    hipFree(d_inst_pot);
    return h_inst_pot[0];
} 

__host__ void cu_potential_energy(int num_atoms,
                                  double3 *pos,
                                  trap_geo params,
                                  wavefunction *psi,
                                  double *potential_energy) { 
#if defined(LOGGING) 
    LOGF(DEBUG, "\nCalculating optimal launch configuration for the potential "
                "energy calculation kernel.\n");
#endif 
    int block_size = 0;
    int min_grid_size = 0;
    int grid_size = 0; 
    hipOccupancyMaxPotentialBlockSize(&min_grid_size,
                                       &block_size,
                                       (const void *) g_potential_energy,
                                       0,
                                       num_atoms); 
    grid_size = (num_atoms + block_size - 1) / block_size; 
#if defined(LOGGING) 
    LOGF(DEBUG, "\nLaunch config set as <<<%i,%i>>>\n", grid_size, block_size);
#endif 

    g_potential_energy<<<grid_size, block_size>>>
                        (num_atoms,
                         pos,
                         params,
                         psi,
                         potential_energy);

    return;
}

__global__ void g_potential_energy(int num_atoms,
                                   double3 *pos,
                                   trap_geo params,
                                   wavefunction *psi,
                                   double *potential_energy) {
    for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
         atom < num_atoms;
         atom += blockDim.x * gridDim.x) {
        potential_energy[atom] = d_potential_energy(pos[atom],
                                                    params,
                                                    psi[atom]);
        if(potential_energy[atom] != potential_energy[atom]) {
            potential_energy[atom] = 0.;
            pos[atom] = make_double3(0., 0., 0.);
            psi[atom] = make_wavefunction(0., 0., 0., 0., true);
        }
    }

    return;
}

__device__ double d_potential_energy(double3 pos,
                                     trap_geo params,
                                     wavefunction psi) {
    hipDoubleComplex potential = make_hipDoubleComplex(0., 0.);
#if defined(SPIN)
    if (psi.isSpinUp) {
        double3 local_B = B(pos,
                            params);
        hipDoubleComplex H[2][2] = {make_hipDoubleComplex(0., 0.)};
        H[0][0] = 0.5*d_gs*d_muB * make_hipDoubleComplex(local_B.z,
                                                        0.);
        H[0][1] = 0.5*d_gs*d_muB * make_hipDoubleComplex(local_B.x,
                                                        -local_B.y);
        H[1][0] = 0.5*d_gs*d_muB * make_hipDoubleComplex(local_B.x,
                                                        local_B.y);
        H[1][1] = 0.5*d_gs*d_muB * make_hipDoubleComplex(-local_B.z,
                                                        0.);
        potential = psi.up*(H[0][0]*hipConj(psi.up) + H[1][0]*hipConj(psi.dn)) +
                    psi.dn*(H[0][1]*hipConj(psi.up) + H[1][1]*hipConj(psi.dn));
    }
#else
    potential = make_hipDoubleComplex(0.5*d_gs*d_muB*norm(B(pos,
                                                           params)),
                                     0.);
#endif

    return hipCreal(potential);
}

__host__ double inst_projection(int num_atoms,
                                double3 *pos,
                                trap_geo params,
                                wavefunction *psi,
                                double *projection) {
    double *h_inst_proj = NULL;
    h_inst_proj = reinterpret_cast<double*>(calloc(1,
                                                  sizeof(double)));
    double *d_inst_proj = NULL;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_inst_proj),
                               sizeof(double)));

    cu_projection(num_atoms,
                  pos,
                  params,
                  psi,
                  projection);
    // Determine temporary device storage requirements
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage,
                                           temp_storage_bytes,
                                           projection,
                                           d_inst_proj,
                                           num_atoms));
    // Allocate temporary storage
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_temp_storage),
                               temp_storage_bytes));
    // Run sum-reduction
    checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage,
                                           temp_storage_bytes,
                                           projection,
                                           d_inst_proj,
                                           num_atoms));
    checkCudaErrors(hipMemcpy(h_inst_proj,
                               d_inst_proj,
                               1.*sizeof(double),
                               hipMemcpyDeviceToHost));
    hipFree(d_temp_storage);
    hipFree(d_inst_proj);

    return h_inst_proj[0];
}

__host__ void cu_projection(int num_atoms,
                            double3 *pos,
                            trap_geo params,
                            wavefunction *psi,
                            double *projection) {
#if defined(LOGGING)
    LOGF(DEBUG, "\nCalculating optimal launch configuration for the projection "
                "calculation kernel.\n");
#endif
    int block_size = 0;
    int min_grid_size = 0;
    int grid_size = 0;
    hipOccupancyMaxPotentialBlockSize(&min_grid_size,
                                       &block_size,
                                       (const void *) g_projection,
                                       0,
                                       num_atoms);
    grid_size = (num_atoms + block_size - 1) / block_size;
#if defined(LOGGING)
    LOGF(DEBUG, "\nLaunch config set as <<<%i,%i>>>\n",
                grid_size, block_size);
#endif

    g_projection<<<grid_size,
                   block_size>>>
                  (num_atoms,
                   pos,
                   params,
                   psi,
                   projection);

    return;
}

__global__ void g_projection(int num_atoms,
                             double3 *pos,
                             trap_geo params,
                             wavefunction *psi,
                             double *projection) {
    for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
         atom < num_atoms;
         atom += blockDim.x * gridDim.x) {
        projection[atom] = d_projection(pos[atom],
                                        params,
                                        &psi[atom]);
        if(projection[atom] != projection[atom]) {
            projection[atom] = 0.;
            pos[atom] = make_double3(0., 0., 0.);
            psi[atom] = make_wavefunction(0., 0., 0., 0., true);
        }
    }

    return;
}

__device__ double d_projection(double3 pos,
                               trap_geo params,
                               wavefunction *psi) {
    hipDoubleComplex P = make_hipDoubleComplex(0., 0.);
    wavefunction l_psi = psi[0];
    if (l_psi.isSpinUp) {
        double3 Bn = unit(B(pos,
                            params));
        P = (Bn.x*l_psi.up - Bn.z*l_psi.dn) * hipConj(l_psi.dn) + 
            (Bn.x*l_psi.dn + Bn.z*l_psi.up) * hipConj(l_psi.up) + 
            (hipCimag(l_psi.dn)*hipCreal(l_psi.up) -
             hipCimag(l_psi.up)*hipCreal(l_psi.dn)) * Bn.y;

        if (hipCreal(P)<0.) {
            //printf("I flipped!\n");
            psi[0].isSpinUp = false;
            P = make_hipDoubleComplex(0., 0.);
        }
    }

    return hipCreal(P);
}

__host__ int inst_is_spin_up(int num_atoms,
                             wavefunction *psi,
                             int *is_spin_up) {
    int *h_inst_spin_up = NULL;
    h_inst_spin_up = reinterpret_cast<int*>(calloc(1,
                                                   sizeof(int)));
    int *d_inst_spin_up = NULL;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_inst_spin_up),
                               sizeof(int)));

    cu_is_spin_up(num_atoms,
                  psi,
                  is_spin_up);
    // Determine temporary device storage requirements
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage,
                                           temp_storage_bytes,
                                           is_spin_up,
                                           d_inst_spin_up,
                                           num_atoms));
    // Allocate temporary storage
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_temp_storage),
                               temp_storage_bytes));
    // Run sum-reduction
    checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage,
                                           temp_storage_bytes,
                                           is_spin_up,
                                           d_inst_spin_up,
                                           num_atoms));
    checkCudaErrors(hipMemcpy(h_inst_spin_up,
                               d_inst_spin_up,
                               1.*sizeof(int),
                               hipMemcpyDeviceToHost));
    hipFree(d_temp_storage);
    hipFree(d_inst_spin_up);

    return h_inst_spin_up[0];
}

__host__ void cu_is_spin_up(int num_atoms,
                            wavefunction *psi,
                            int *is_spin_up) {
#if defined(LOGGING)
    LOGF(DEBUG, "\nCalculating optimal launch configuration for the is_spin_up "
                "calculation kernel.\n");
#endif
    int block_size = 0;
    int min_grid_size = 0;
    int grid_size = 0;
    hipOccupancyMaxPotentialBlockSize(&min_grid_size,
                                       &block_size,
                                       (const void *) g_is_spin_up,
                                       0,
                                       num_atoms);
    grid_size = (num_atoms + block_size - 1) / block_size;
#if defined(LOGGING)
    LOGF(DEBUG, "\nLaunch config set as <<<%i,%i>>>\n",
                grid_size, block_size);
#endif

    g_is_spin_up<<<grid_size,
                   block_size>>>
                  (num_atoms,
                   psi,
                   is_spin_up);

    return;
}

__global__ void g_is_spin_up(int num_atoms,
                             wavefunction *psi,
                             int *is_spin_up) {
    for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
         atom < num_atoms;
         atom += blockDim.x * gridDim.x) {
        is_spin_up[atom] = d_is_spin_up(psi[atom]);
        if(is_spin_up[atom] != is_spin_up[atom]) {
            is_spin_up[atom] = 0;
            psi[atom] = make_wavefunction(0., 0., 0., 0., true);
        }
    }

    return;
}

__device__ int d_is_spin_up(wavefunction psi) {
    int is_spin_up = 0;
    if (psi.isSpinUp) is_spin_up = 1;

    return is_spin_up;
}

