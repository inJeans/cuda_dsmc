#include "hip/hip_runtime.h"
/** \file
 *  \brief Unit tests for the collisions file
 *
 *  More detailed description
 *  Copyright 2015 Christopher Watkins
 */

#include "ehrenfest_test.cuh"

double fractional_tol = 0.05; 

SCENARIO("[DEVICE] Execute a full ehrenfest simulation", "[d-ehrenfest]") {
    GIVEN("A thermally distributed intial distribution.") {

        // Initialise trapping parameters
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the trapping parameters.");
#endif
#if defined(IP)  // Ioffe Pritchard trap
        trap_geo trap_parameters;
        trap_parameters.B0 = 0.01;
        trap_parameters.dB = 20.;
        trap_parameters.ddB = 40000.;
#else  // Quadrupole trap
        trap_geo trap_parameters;
        trap_parameters.Bz = 2.0;
        trap_parameters.B0 = 0.;
#endif

        // Initialise computational parameters
        int num_atoms = 1e1;
        FN = 10;
        double dt = 1.e-6;
        int num_time_steps = 10;
        double init_temp = 20.e-6;

        // Initialise grid parameters
        k_num_cells = make_int3(35, 35, 35);
        total_num_cells = k_num_cells.x*k_num_cells.y*k_num_cells.z;

    // Initialise rng
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the rng state array.");
        LOGF(DEBUG, "\nAllocating %i hiprandState elements on the device.",
             num_atoms);
#endif
        hiprandState *state;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&state),
                                   num_atoms*sizeof(hiprandState)));
        initialise_rng_states(num_atoms,
                              state,
                              false);

        // Initialise atom_id
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the atom_id array.");
        LOGF(DEBUG, "\nAllocating %i int elements on the device.",
             num_atoms);
#endif
        int *atom_id;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&atom_id),
                                   num_atoms*sizeof(int)));
        initialise_atom_id(num_atoms,
                           atom_id);

        // Initialise cell_id
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the cell_id array.");
        LOGF(DEBUG, "\nAllocating %i int elements on the device.",
             num_atoms);
#endif
        int *cell_id;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&cell_id),
                                   num_atoms*sizeof(int)));
        checkCudaErrors(hipMemset(cell_id,
                                   0,
                                   num_atoms*sizeof(int)));

        // Initialise cell_start_end
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the cell_start_end array.");
        LOGF(DEBUG, "\nAllocating %i int2 elements on the device.",
             total_num_cells+1);
#endif
        int2 *cell_start_end;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&cell_start_end),
                                   (total_num_cells+1)*sizeof(int2)));
        checkCudaErrors(hipMemset(cell_start_end,
                                   -1,
                                   (total_num_cells+1)*sizeof(int2)));

        // Initialise cell_num_atoms
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the cell_num_atoms array.");
        LOGF(DEBUG, "\nAllocating %i int elements on the device.",
             total_num_cells+1);
#endif
        int *cell_num_atoms;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&cell_num_atoms),
                                   (total_num_cells+1)*sizeof(int)));
        checkCudaErrors(hipMemset(cell_num_atoms,
                                   0,
                                   (total_num_cells+1)*sizeof(int)));

        // Initialise cell_cumulative_num_atoms
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the cell_cumulative_num_atoms array.");
        LOGF(DEBUG, "\nAllocating %i int elements on the device.",
             total_num_cells+1);
#endif
        int *cell_cumulative_num_atoms;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&cell_cumulative_num_atoms),
                                   (total_num_cells+1)*sizeof(int)));
        checkCudaErrors(hipMemset(cell_cumulative_num_atoms,
                                   0,
                                   (total_num_cells+1)*sizeof(int)));

        // Initialise collision_count
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the collision_count array.");
        LOGF(DEBUG, "\nAllocating %i int elements on the device.",
             total_num_cells);
#endif
        int *collision_count;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&collision_count),
                                   (total_num_cells)*sizeof(int)));
        checkCudaErrors(hipMemset(collision_count,
                                   0,
                                   (total_num_cells)*sizeof(int)));

        // Initialise collision_remainder
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the collision_remainder array.");
        LOGF(DEBUG, "\nAllocating %i int elements on the device.",
             total_num_cells);
#endif
        double *collision_remainder;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&collision_remainder),
                                   (total_num_cells)*sizeof(double)));
        zero_elements<<<total_num_cells,1>>>(total_num_cells,
                                             collision_remainder);

        // Initialise sig_vr_max
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the sig_vr_max array.");
        LOGF(DEBUG, "\nAllocating %i int elements on the device.",
             total_num_cells);
#endif
        double *sig_vr_max;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&sig_vr_max),
                                   (total_num_cells)*sizeof(double)));
        zero_elements<<<total_num_cells,1>>>(total_num_cells,
                                             sig_vr_max);

    // Initialise velocities
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the velocity array.");
        LOGF(DEBUG, "\nAllocating %i double3 elements on the device.",
             num_atoms);
#endif
        double3 *vel;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&vel),
                                   num_atoms*sizeof(double3)));

        // Generate velocity distribution
        generate_thermal_velocities(num_atoms,
                                    init_temp,
                                    state,
                                    vel);

        // Initialise positions
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the position array.");
        LOGF(DEBUG, "\nAllocating %i double3 elements on the device.",
             num_atoms);
#endif
        double3 *pos;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&pos),
                                   num_atoms*sizeof(double3)));

        // Generate position distribution
        generate_thermal_positions(num_atoms,
                                   init_temp,
                                   trap_parameters,
                                   state,
                                   pos);

        // Initialise wavefunction
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the wavefunction array.");
        LOGF(DEBUG, "\nAllocating %i wavefunction elements on the device.",
             num_atoms);
#endif  // Logging
        wavefunction *psi;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&psi),
                                   num_atoms*sizeof(wavefunction)));

        // Generate wavefunction
        generate_aligned_spins(num_atoms,
                               trap_parameters,
                               pos,
                               psi);

        // Initialise accelerations
#if defined(LOGGING)
        LOGF(INFO, "\nInitialising the acceleration array.");
        LOGF(DEBUG, "\nAllocating %i double3 elements on the device.",
             num_atoms);
#endif
        double3 *acc;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&acc),
                                   num_atoms*sizeof(double3)));

        // Generate accelerations
        update_accelerations(num_atoms,
                             trap_parameters,
                             pos,
                             acc,
                             psi);

#if defined(LOGGING)
        LOGF(DEBUG, "\nCreating the cuBLAS handle.\n");
#endif
        hipblasHandle_t cublas_handle;
        checkCudaErrors(hipblasCreate(&cublas_handle));

        // Set up global grid parameters
        initialise_grid_params(num_atoms,
                               cublas_handle,
                               pos);

        // Evolve many time step
#if defined(LOGGING)
        LOGF(INFO, "\nEvolving distribution for %i time steps.", num_time_steps);
#endif
        for (int t = 0; t < num_time_steps; ++t) {
            velocity_verlet_update(num_atoms,
                                   dt,
                                   trap_parameters,
                                   cublas_handle,
                                   pos,
                                   vel,
                                   acc,
                                   psi);

            collide_atoms(num_atoms,
                          total_num_cells,
                          dt,
                          pos,
                          vel,
                          state,
                          sig_vr_max,
                          cell_id,
                          atom_id,
                          cell_start_end,
                          cell_num_atoms,
                          cell_cumulative_num_atoms,
                          collision_remainder,
                          collision_count);

            progress_bar(t,
                         num_time_steps);
        }

#if defined(LOGGING)
        LOGF(DEBUG, "\nDestroying the cuBLAS handle.\n");
#endif
        checkCudaErrors(hipblasDestroy(cublas_handle));

#if defined(LOGGING)
        LOGF(INFO, "\nCleaning up device memory.");
#endif
        hipFree(state);
        hipFree(atom_id);
        hipFree(cell_id);
        hipFree(cell_start_end);
        hipFree(cell_num_atoms);
        hipFree(cell_cumulative_num_atoms);
        hipFree(collision_count);
        hipFree(collision_remainder);
        hipFree(sig_vr_max);
        hipFree(vel);
        hipFree(pos);
        hipFree(acc);
        hipFree(psi);

    }
}